#include "hip/hip_runtime.h"
#include "bfs_simple.cuh"

using namespace std;

#define DEBUG(x)
#define N_THREADS_PER_BLOCK (1 << 5)



__global__
void initializeDeviceArray(int n, int *d_arr, int value, int start_index) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid == start_index) {
		d_arr[start_index] = 0;
	}
	else if (tid < n) {
		d_arr[tid] = value;
	}
}


__global__
void printDeviceArray(int *d_arr, int n) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		printf("d_arr[%i] = %i \n", tid, d_arr[tid]);
	}
}


/*
 * Given a graph and a current queue computes next vertices (vertex frontiers) to traverse.
 */
__global__
void computeNextQueue(int *adjacencyList, int *edgesOffset, int *edgesSize, int *distance,
		int queueSize, int *currentQueue, int *nextQueueSize, int *nextQueue, int level) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;  // thread id
	if (tid < queueSize) {  // visit all vertexes in a queue in parallel
		int current = currentQueue[tid];
		for (int i = edgesOffset[current]; i < edgesOffset[current] + edgesSize[current]; ++i) {
			int v = adjacencyList[i];
			if (distance[v] == INT_MAX) {
				distance[v] = level + 1;
				int position = atomicAdd(nextQueueSize, 1);
				nextQueue[position] = v;
			}
		}
	}
}


void bfsGPU(int start, Graph &G, vector<int> &distance, vector<bool> &visited) {

	const int n_blocks = (G.numVertices + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK;

	// Initialization of GPU variables
	int *d_adjacencyList;
	int *d_edgesOffset;
	int *d_edgesSize;
	int *d_firstQueue;
	int *d_secondQueue;
	int *d_nextQueueSize;
	int *d_distance; // output

	// Initialization of CPU variables
	int currentQueueSize = 1;
	const int NEXT_QUEUE_SIZE = 0;
	int level = 0;

	// Allocation on device
	const int size = G.numVertices * sizeof(int);
	const int adjacencySize = G.adjacencyList.size() * sizeof(int);
	hipMalloc((void **)&d_adjacencyList, adjacencySize);
	hipMalloc((void **)&d_edgesOffset, size);
	hipMalloc((void **)&d_edgesSize, size);
	hipMalloc((void **)&d_firstQueue, size);
	hipMalloc((void **)&d_secondQueue, size);
	hipMalloc((void **)&d_distance, size);
	hipMalloc((void **)&d_nextQueueSize, sizeof(int));

	// Copy inputs to device
	hipMemcpy(d_adjacencyList, &G.adjacencyList[0], adjacencySize, hipMemcpyHostToDevice);
	hipMemcpy(d_edgesOffset, &G.edgesOffset[0], size, hipMemcpyHostToDevice);
	hipMemcpy(d_edgesSize, &G.edgesSize[0], size, hipMemcpyHostToDevice);
	hipMemcpy(d_nextQueueSize, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_firstQueue, &start, sizeof(int), hipMemcpyHostToDevice);
//	initializeDeviceArray<<<n_blocks, N_THREADS_PER_BLOCK>>> (G.numVertices, d_distance, INT_MAX, start); // FOR SOME REASON USING THIS KERNEL DOESNT WORK
//	hipDeviceSynchronize();
	distance = vector<int> (G.numVertices, INT_MAX);
	distance[start] = 0;
	auto startTime = chrono::steady_clock::now();
	hipMemcpy(d_distance, distance.data(), size, hipMemcpyHostToDevice);

	while (currentQueueSize > 0) {
		int *d_currentQueue;
		int *d_nextQueue;
		if (level % 2 == 0) {
			d_currentQueue = d_firstQueue;
			d_nextQueue = d_secondQueue;
		}
		else {
			d_currentQueue = d_secondQueue;
			d_nextQueue = d_firstQueue;
		}
		computeNextQueue<<<n_blocks, N_THREADS_PER_BLOCK>>> (d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance,
				currentQueueSize, d_currentQueue, d_nextQueueSize, d_nextQueue, level);
		hipDeviceSynchronize();
		++level;
		hipMemcpy(&currentQueueSize, d_nextQueueSize, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(d_nextQueueSize, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
	}

	hipMemcpy(&distance[0], d_distance, size, hipMemcpyDeviceToHost);
	auto endTime = std::chrono::steady_clock::now();
	auto duration = chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count();
	printf("Elapsed time for naive linear GPU implementation (without copying graph) : %li ms.\n", duration);

	// Cleanup
	hipFree(d_adjacencyList);
	hipFree(d_edgesOffset);
	hipFree(d_edgesSize);
	hipFree(d_firstQueue);
	hipFree(d_secondQueue);
	hipFree(d_distance);
}
