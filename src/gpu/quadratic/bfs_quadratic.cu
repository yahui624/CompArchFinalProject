#include "hip/hip_runtime.h"
#include "bfs_quadratic.cuh"

using namespace std;

#define DEBUG(x)
#define N_THREADS_PER_BLOCK (1 << 2)



__global__
void compute_next_layer_distance(int n, int *adjacencyList, int *edgesOffset, int *edgesSize, int *distance, int level, bool *done) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		if (distance[tid] == level) {
			for (int i = edgesOffset[tid]; i < edgesOffset[tid] + edgesSize[tid]; ++i) {
				int v = adjacencyList[i];
				if (distance[v] == INT_MAX) {
					*done = false;
					distance[v] = level + 1;
				}
			}
		}
	}
}


// Assumes that distance is a vector of all INT_MAX (except at start position)
void bfsGPUQuadratic(int start, Graph &G, vector<int> &distance, vector<bool> &visited) {

	const int n_blocks = (G.numVertices + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK;

	// Initialization of GPU variables
	int *d_adjacencyList;
	int *d_edgesOffset;
	int *d_edgesSize;
	int *d_distance; // output
	bool done = false;
	const bool true_value = true;

	// Initialization of CPU variables
	bool *d_done;
	int level = 0;

	// Allocation on device
	const int size = G.numVertices * sizeof(int);
	const int adjacencySize = G.adjacencyList.size() * sizeof(int);
	hipMalloc((void **)&d_adjacencyList, adjacencySize);
	hipMalloc((void **)&d_edgesOffset, size);
	hipMalloc((void **)&d_edgesSize, size);
	hipMalloc((void **)&d_distance, size);
	hipMalloc((void **)&d_done, sizeof(bool));  // malloc of single value is also important


	// Copy inputs to device
	hipMemcpy(d_adjacencyList, &G.adjacencyList[0], adjacencySize, hipMemcpyHostToDevice);
	hipMemcpy(d_edgesOffset, &G.edgesOffset[0], size, hipMemcpyHostToDevice);
	hipMemcpy(d_edgesSize, &G.edgesSize[0], size, hipMemcpyHostToDevice);
	distance[start] = 0;
	hipMemcpy(d_distance, distance.data(), size, hipMemcpyHostToDevice);

	while (!done) {
		hipMemcpy(d_done, &true_value, sizeof(bool), hipMemcpyHostToDevice);
		compute_next_layer_distance <<<n_blocks, N_THREADS_PER_BLOCK>>> (G.numVertices, d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance, level, d_done);
		hipDeviceSynchronize();
		hipMemcpy(&done, d_done, sizeof(bool), hipMemcpyDeviceToHost);
		++level;
		if (level > G.numVertices) {
			cerr << "Number of iterations exceeded number of vertices!" << endl;
			break;
		}
	}

	// Copying output back to host
	hipMemcpy(&distance[0], d_distance, size, hipMemcpyDeviceToHost);

	// Cleanup
	hipFree(d_adjacencyList);
	hipFree(d_edgesOffset);
	hipFree(d_edgesSize);
	hipFree(d_distance);
}
